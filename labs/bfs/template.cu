#include "hip/hip_runtime.h"
#include <atomic>
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512
#define WARP_SIZE 32
// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 4096

// Number of warp queues per block
#define NUM_WARP_QUEUES 8
#define NUM_WARPS (BLOCK_SIZE / WARP_SIZE)
// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY (BQ_CAPACITY / NUM_WARP_QUEUES)

/******************************************************************************
 GPU kernels
*******************************************************************************/

__global__ void gpu_global_queueing_kernel(unsigned int *nodePtrs,
                                          unsigned int *nodeNeighbors,
                                          unsigned int *nodeVisited,
                                          unsigned int *currLevelNodes,
                                          unsigned int *nextLevelNodes,
                                          unsigned int *numCurrLevelNodes,
                                          unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE
  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If neighbor hasn't been visited yet
  // Add neighbor to global queue
  unsigned int nCurrLevel = *numCurrLevelNodes;
  int tId = blockDim.x * blockIdx.x + threadIdx.x;
  if (tId == 0) {
    *numNextLevelNodes = 0;
  }
  __syncthreads();
  if (tId < nCurrLevel) {
    unsigned int node = currLevelNodes[tId];
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1]; ++nbrIdx) {
      unsigned int neighborID = nodeNeighbors[nbrIdx];
      unsigned int visit = nodeVisited[neighborID];
      if (!visit) {
        if (atomicCAS(&nodeVisited[neighborID], visit, 1) == visit) {
          int tail = atomicAdd(numNextLevelNodes, 1);  // get the current tail
          nextLevelNodes[tail] = neighborID;
        }
      }
    }
  }
}

__global__ void gpu_block_queueing_kernel(unsigned int *nodePtrs,
                                         unsigned int *nodeNeighbors,
                                         unsigned int *nodeVisited,
                                         unsigned int *currLevelNodes,
                                         unsigned int *nextLevelNodes,
                                         unsigned int *numCurrLevelNodes,
                                         unsigned int *numNextLevelNodes) {
  // INSERT KERNEL CODE HERE

  // Initialize shared memory queue (size should be BQ_CAPACITY)
  __shared__ unsigned int blockQueue[BQ_CAPACITY];
  __shared__ unsigned int blocktail, globaltail;
  unsigned int nCurrLevel = *numCurrLevelNodes;
  int tId = blockDim.x * blockIdx.x + threadIdx.x;
  if (threadIdx.x == 0) {
    blocktail = 0;
  }
  if (tId == 0) {
    *numNextLevelNodes = 0;
  }
  __syncthreads();
  if (tId < nCurrLevel) {
    unsigned int node = currLevelNodes[tId];
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1]; ++nbrIdx) {
      unsigned int neighborID = nodeNeighbors[nbrIdx];
      unsigned int visit = nodeVisited[neighborID];
      if (!visit) {
        if (atomicCAS(&nodeVisited[neighborID], visit, 1) == visit) {
          int tail = atomicAdd(&blocktail, 1);  // get the current tail
          if(tail < BQ_CAPACITY){
            blockQueue[tail] = neighborID;
          }
          else{
            tail = BQ_CAPACITY;
            nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighborID;
          }
        }
      }
    }
  }
  __syncthreads();
  if(threadIdx.x == 0)
    globaltail = atomicAdd(numNextLevelNodes, blocktail);
  __syncthreads();
  // int perthread = ceil(blocktail / blockDim.x);
  for (int i = threadIdx.x; i < blocktail; i += blockDim.x) {
    nextLevelNodes[globaltail + i] = blockQueue[i];
  }
  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If neighbor hasn't been visited yet
  // Add neighbor to block queue
  // If full, add neighbor to global queue

  // Allocate space for block queue to go into global queue

  // Store block queue in global queue
}

__global__ void gpu_warp_queueing_kernel(unsigned int *nodePtrs,
                                        unsigned int *nodeNeighbors,
                                        unsigned int *nodeVisited,
                                        unsigned int *currLevelNodes,
                                        unsigned int *nextLevelNodes,
                                        unsigned int *numCurrLevelNodes,
                                        unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE

  __shared__ unsigned int warpQueue[WQ_CAPACITY][NUM_WARP_QUEUES];
  __shared__ unsigned int blockQueue[BQ_CAPACITY];
  __shared__ unsigned int warpQueueTail[NUM_WARP_QUEUES];
  __shared__ unsigned int warpQueuePrefix[NUM_WARP_QUEUES];
  __shared__ unsigned int blockQueueTail, blockToGlobalQueueTail;
  int tId = blockDim.x * blockIdx.x + threadIdx.x;
  if(threadIdx.x / NUM_WARP_QUEUES == 0){
    warpQueueTail[threadIdx.x] = 0;
    if(threadIdx.x == 0){
      blockQueueTail = 0;
      if(tId == 0)
        *numNextLevelNodes = 0;
    }
  }
  __syncthreads();
  unsigned int nCurrLevel = *numCurrLevelNodes;
  unsigned int warpId = threadIdx.x % NUM_WARP_QUEUES;

  if (tId < nCurrLevel) {
    unsigned int node = currLevelNodes[tId];
    for (unsigned int nbrIdx = nodePtrs[node]; nbrIdx < nodePtrs[node + 1]; ++nbrIdx) {
      unsigned int neighborID = nodeNeighbors[nbrIdx];
      if (!atomicExch(&nodeVisited[neighborID], 1)) {
        int tail = atomicAdd(&warpQueueTail[warpId], 1);  // get the current tail
        if(tail < WQ_CAPACITY){
          warpQueue[tail][warpId] = neighborID;
        }
        else{
          warpQueueTail[warpId] = WQ_CAPACITY;
          int btail = atomicAdd(&(blockQueueTail), 1);
          if (btail < BQ_CAPACITY) {
            blockQueue[btail] = neighborID;
          } else {
            blockQueueTail = BQ_CAPACITY;
            nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighborID;
          }
        }
      }
    }
  }

  __syncthreads();
  if(threadIdx.x == 0){
    warpQueuePrefix[0] = blockQueueTail;
    for(unsigned int i = 1;i < NUM_WARP_QUEUES;i++)
      warpQueuePrefix[i] = warpQueuePrefix[i - 1] + warpQueueTail[i - 1];
  }

  __syncthreads();
  if(threadIdx.x == 0){
    if(warpQueuePrefix[NUM_WARP_QUEUES - 1] + warpQueueTail[NUM_WARP_QUEUES - 1] < BQ_CAPACITY)
      blockQueueTail = warpQueuePrefix[NUM_WARP_QUEUES - 1] + warpQueueTail[NUM_WARP_QUEUES - 1];
    else
      blockQueueTail = BQ_CAPACITY; 
    blockToGlobalQueueTail = atomicAdd(numNextLevelNodes, blockQueueTail);
  }
  const unsigned int warpIdx = threadIdx.x / WARP_SIZE;
  if (warpIdx < NUM_WARP_QUEUES) {
    for(unsigned int i = threadIdx.x % WARP_SIZE; i < warpQueueTail[warpIdx];i += WARP_SIZE){
      const unsigned int warpToBlockQueueIdx = warpQueuePrefix[warpIdx] + i;
      if(warpToBlockQueueIdx < BQ_CAPACITY)
        blockQueue[warpToBlockQueueIdx] = warpQueue[i][warpIdx];
      else
        nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = warpQueue[i][warpIdx];
    }
  }
  __syncthreads();
  for(unsigned int i = threadIdx.x;i < blockQueueTail;i += blockDim.x)
    nextLevelNodes[blockToGlobalQueueTail + i] = blockQueue[i];


  
  // This version uses NUM_WARP_QUEUES warp queues of capacity 
  // WQ_CAPACITY.  Be sure to interleave them as discussed in lecture.  

  // Don't forget that you also need a block queue of capacity BQ_CAPACITY.

  // Initialize shared memory queues (warp and block)

  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If neighbor hasn't been visited yet
  // Add neighbor to the queue
  // If full, add neighbor to block queue
  // If full, add neighbor to global queue

  // Allocate space for warp queue to go into block queue

  // Store warp queues in block queue (use one warp or one thread per queue)
  // Add any nodes that don't fit (remember, space was allocated above)
  //    to the global queue

  // Saturate block queue counter (too large if warp queues overflowed)
  // Allocate space for block queue to go into global queue

  // Store block queue in global queue
}

/******************************************************************************
 Functions
*******************************************************************************/
// DON NOT MODIFY THESE FUNCTIONS!

void gpu_global_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                        unsigned int *nodeVisited, unsigned int *currLevelNodes,
                        unsigned int *nextLevelNodes,
                        unsigned int *numCurrLevelNodes,
                        unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queueing_kernel <<<numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_block_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                       unsigned int *nodeVisited, unsigned int *currLevelNodes,
                       unsigned int *nextLevelNodes,
                       unsigned int *numCurrLevelNodes,
                       unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queueing_kernel <<<numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_warp_queueing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                      unsigned int *nodeVisited, unsigned int *currLevelNodes,
                      unsigned int *nextLevelNodes,
                      unsigned int *numCurrLevelNodes,
                      unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queueing_kernel <<<numBlocks, BLOCK_SIZE>>>(nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}
